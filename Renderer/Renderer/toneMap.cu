#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "SampleRenderer.h"

using namespace osc;

namespace osc {

  inline __device__ float4 sqrt(float4 f)
  {
    return make_float4(sqrtf(f.x),
                       sqrtf(f.y),
                       sqrtf(f.z),
                       sqrtf(f.w));
  }
  inline __device__ float  clampf(float f) { return min(1.f,max(0.f,f)); }
  inline __device__ float4 clamp(float4 f)
  {
    return make_float4(clampf(f.x),
                       clampf(f.y),
                       clampf(f.z),
                       clampf(f.w));
  }
  
  /*! runs a cuda kernel that performs gamma correction and float4-to-rgba conversion */
  __global__ void computeFinalPixelColorsKernel(uint32_t *finalColorBuffer,
                                                float4   *denoisedBuffer,
                                                vec2i     size)
  {
    int pixelX = threadIdx.x + blockIdx.x*blockDim.x;
    int pixelY = threadIdx.y + blockIdx.y*blockDim.y;
    if (pixelX >= size.x) return;
    if (pixelY >= size.y) return;

    int pixelID = pixelX + size.x*pixelY;

    float4 f4 = denoisedBuffer[pixelID];
    f4 = clamp(sqrt(f4));
    uint32_t rgba = 0;
    rgba |= (uint32_t)(f4.x * 255.9f) <<  0;
    rgba |= (uint32_t)(f4.y * 255.9f) <<  8;
    rgba |= (uint32_t)(f4.z * 255.9f) << 16;
    rgba |= (uint32_t)255             << 24;
    finalColorBuffer[pixelID] = rgba;
  }

  void SampleRenderer::computeFinalPixelColors()
  {
    vec2i fbSize = launchParams.frame.size;
    vec2i blockSize = 32;
    vec2i numBlocks = divRoundUp(fbSize,blockSize);
    computeFinalPixelColorsKernel
      <<<dim3(numBlocks.x,numBlocks.y),dim3(blockSize.x,blockSize.y)>>>
      ((uint32_t*)finalColorBuffer.d_pointer(),
       (float4*)denoisedBuffer.d_pointer(),
       fbSize);
  }
  
} // ::osc
